#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"

extern "C" {
#include "convolutional_layer.h"
#include "deconvolutional_layer.h"
#include "batchnorm_layer.h"
#include "gemm.h"
#include "blas.h"
#include "im2col.h"
#include "col2im.h"
#include "utils.h"
#include "hip/hip_runtime.h"
}

extern "C" void forward_deconvolutional_layer_gpu(layer l, network_state state)
{
    int i;
    int out_h = l.out_h;
    int out_w = l.out_w;
    int size = out_h*out_w;

    int m = l.size*l.size*l.n;
    int n = l.h*l.w;
    int k = l.c;

    fill_ongpu(l.outputs*l.batch, 0, l.output_gpu, 1);

    for(i = 0; i < l.batch; ++i){
        float *a = l.weights_gpu;
        float *b = state.input + i*l.c*l.h*l.w;
        float *c = state.workspace;

        gemm_ongpu(1,0,m,n,k,1,a,m,b,n,0,c,n);

        col2im_ongpu(c, l.n, out_h, out_w, l.size, l.stride, l.pad, l.output_gpu+i*l.n*size);
    }
    if (l.batch_normalize) {
        forward_batchnorm_layer_gpu(l, state);
    } else {
        add_bias_gpu(l.output_gpu, l.biases_gpu, l.batch, l.n, l.out_w*l.out_h);
    }
    activate_array_ongpu(l.output_gpu, l.batch*l.n*size, l.activation);
}

extern "C" void backward_deconvolutional_layer_gpu(layer l, network_state state)
{
    int out_h = l.out_h;
    int out_w = l.out_w;
    int size = out_h*out_w;
    int i;

    gradient_array_ongpu(l.output_gpu, l.outputs*l.batch, l.activation, l.delta_gpu);

    if(l.batch_normalize){
        backward_batchnorm_layer_gpu(l, state);
    } else {
        backward_bias_gpu(l.bias_updates_gpu, l.delta_gpu, l.batch, l.n, l.out_w*l.out_h);
    }

    //if(state.delta) memset(state.delta, 0, l.batch*l.h*l.w*l.c*sizeof(float));

    for(i = 0; i < l.batch; ++i){
        int m = l.c;
        int n = l.size*l.size*l.n;
        int k = l.h*l.w;

        float *a = state.input + i*m*n;
        float *b = state.workspace;
        float *c = l.weight_updates_gpu;

        im2col_ongpu(l.delta_gpu + i*l.n*size, l.n, out_h, out_w, 
                l.size, l.stride, l.pad, b);
        gemm_ongpu(0,1,m,n,k,1,a,k,b,k,1,c,n);

        if(state.delta){
            int m = l.c;
            int n = l.h*l.w;
            int k = l.size*l.size*l.n;

            float *a = l.weights_gpu;
            float *b = state.workspace;
            float *c = state.delta + i*n*m;

            gemm_ongpu(0,0,m,n,k,1,a,k,b,n,1,c,n);
        }
    }
}

extern "C" void pull_deconvolutional_layer(layer l)
{
    cuda_pull_array(l.weights_gpu, l.weights, l.c*l.n*l.size*l.size);
    cuda_pull_array(l.biases_gpu, l.biases, l.n);
    cuda_pull_array(l.weight_updates_gpu, l.weight_updates, l.c*l.n*l.size*l.size);
    cuda_pull_array(l.bias_updates_gpu, l.bias_updates, l.n);
    if (l.batch_normalize){
        cuda_pull_array(l.scales_gpu, l.scales, l.n);
        cuda_pull_array(l.rolling_mean_gpu, l.rolling_mean, l.n);
        cuda_pull_array(l.rolling_variance_gpu, l.rolling_variance, l.n);
    }
}

extern "C" void push_deconvolutional_layer(layer l)
{
    cuda_push_array(l.weights_gpu, l.weights, l.c*l.n*l.size*l.size);
    cuda_push_array(l.biases_gpu, l.biases, l.n);
    cuda_push_array(l.weight_updates_gpu, l.weight_updates, l.c*l.n*l.size*l.size);
    cuda_push_array(l.bias_updates_gpu, l.bias_updates, l.n);
    if (l.batch_normalize){
        cuda_push_array(l.scales_gpu, l.scales, l.n);
        cuda_push_array(l.rolling_mean_gpu, l.rolling_mean, l.n);
        cuda_push_array(l.rolling_variance_gpu, l.rolling_variance, l.n);
    }
}

void update_deconvolutional_layer_gpu(layer l, int batch, float learning_rate, float momentum, float decay)
{
    int size = l.size*l.size*l.c*l.n;
    axpy_ongpu(l.n, learning_rate/batch, l.bias_updates_gpu, 1, l.biases_gpu, 1);
    scal_ongpu(l.n, momentum, l.bias_updates_gpu, 1);

    if(l.scales_gpu){
        axpy_ongpu(l.n, learning_rate/batch, l.scale_updates_gpu, 1, l.scales_gpu, 1);
        scal_ongpu(l.n, momentum, l.scale_updates_gpu, 1);
    }

    axpy_ongpu(size, -decay*batch, l.weights_gpu, 1, l.weight_updates_gpu, 1);
    axpy_ongpu(size, learning_rate/batch, l.weight_updates_gpu, 1, l.weights_gpu, 1);
    scal_ongpu(size, momentum, l.weight_updates_gpu, 1);
}

